#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void AddIntsCuda(int* a, int* b) {
	a[0] = a[0] + b[0];
}

int main() {
	int a = 5, b = 9;
	int* d_a, * d_b;
	if (hipMalloc(&d_a, sizeof(int)) == hipSuccess) {
		cout << "successfull allocation of a" << endl;
	}
	if (hipMalloc(&d_b, sizeof(int)) == hipSuccess) {
		cout << "successfull allocation of b" << endl;
	}


	hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

	AddIntsCuda<<<1, 1 >>> (d_a, d_b);

	hipMemcpy(&a, d_a, sizeof(int), hipMemcpyDeviceToHost);

	cout << "addition result: " << a << endl;

	hipFree(d_a);
	hipFree(d_b);

	return 0;
}
